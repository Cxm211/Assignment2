#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <ctype.h>
#include <errno.h>
#include <pthread.h>
#include "cuda_thread.h"
#include "util.h"
#include "exporter.h"
#include "settings.h"

//  including the "dead faction": 0
#define MAX_FACTIONS 10

// this macro is here to make the code slightly more readable, not because it can be safely changed to
// any integer value; changing this to a non-zero value may break the code
#define DEAD_FACTION 0

#ifndef MAX_THREAD_NUM
#define MAX_THREAD_NUM 600000
#endif
/**
 * Specifies the number(s) of live neighbors of the same faction required for a dead cell to become alive.
 */
__device__  int death[MAX_THREAD_NUM];

__device__ bool isBirthable(int n) {
    return n == 3;
}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a live cell to remain alive.
 */
__device__ bool isSurvivable(int n) {
    return n == 2 || n == 3;
}

/**
 * Specifies the number of live neighbors of a different faction required for a live cell to die due to fighting.
 */
__device__ bool willFight(int n) {
    return n > 0;
}


void GlobalsetValueAt(int *grid, int nRows, int nCols, int row, int col, int val) {
    if (row < 0 || row >= nRows || col < 0 || col >= nCols) {
        return;
    }

    *(grid + (row * nCols) + col) = val;
}


int GlobalgetValueAt(const int *grid, int nRows, int nCols, int row, int col) {
    if (row < 0 || row >= nRows || col < 0 || col >= nCols) {
        return -1;
    }

    return *(grid + (row * nCols) + col);
}

/**
 * Computes and returns the next state of the cell specified by row and col based on currWorld and invaders. Sets *diedDueToFighting to
 * true if this cell should count towards the death toll due to fighting.
 *
 * invaders can be NULL if there are no invaders.
 */

__device__ int getNextState(const int *currWorld, const int *invaders, int nRows, int nCols, int row, int col,
                            bool *diedDueToFighting) {
    // we'll explicitly set if it was death due to fighting
    *diedDueToFighting = false;
    // faction of this cell
    int cellFaction = getValueAt(currWorld, nRows, nCols, row, col);

    // did someone just get landed on?
    if (invaders != NULL && getValueAt(invaders, nRows, nCols, row, col) != DEAD_FACTION) {
        *diedDueToFighting = cellFaction != DEAD_FACTION;
        return getValueAt(invaders, nRows, nCols, row, col);
    }

    // tracks count of each faction adjacent to this cell
    int neighborCounts[MAX_FACTIONS];
    for (int i = 0; i < MAX_FACTIONS; i++) {
        neighborCounts[i] = 0;
    }
    // memset(neighborCounts, 0, MAX_FACTIONS * sizeof(int));

    // count neighbors (and self)
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int faction = getValueAt(currWorld, nRows, nCols, row + dy, col + dx);
            if (faction >= DEAD_FACTION) {
                neighborCounts[faction]++;
                //printf("NEIGHBOR: %d", neighborCounts[faction]);
            }
        }
    }

    // we counted this cell as its "neighbor"; adjust for this
    neighborCounts[cellFaction]--;
//    for(int i = 0; i < MAX_FACTIONS; i++){
//        printf("N: %d", neighborCounts[i]);
//    }
    if (cellFaction == DEAD_FACTION) {
        // this is a dead cell; we need to see if a birth is possible:
        // need exactly 3 of a single faction; we don't care about other factions

        // by default, no birth
        int newFaction = DEAD_FACTION;

        // start at 1 because we ignore dead neighbors
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++) {
            int count = neighborCounts[faction];
//            printf("COUNT: %d", count);
            if (isBirthable(count)) {
                newFaction = faction;
            }
        }

        return newFaction;
    } else {
        /**
         * this is a live cell; we follow the usual rules:
         * Death (fighting): > 0 hostile neighbor
         * Death (underpopulation): < 2 friendly neighbors and 0 hostile neighbors
         * Death (overpopulation): > 3 friendly neighbors and 0 hostile neighbors
         * Survival: 2 or 3 friendly neighbors and 0 hostile neighbors
         */

        int hostileCount = 0;
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++) {
            if (faction == cellFaction) {
                continue;
            }
            hostileCount += neighborCounts[faction];
        }

        if (willFight(hostileCount)) {
            *diedDueToFighting = true;
            return DEAD_FACTION;
        }

        int friendlyCount = neighborCounts[cellFaction];
        if (!isSurvivable(friendlyCount)) {
            return DEAD_FACTION;
        }

        return cellFaction;
    }
}


__global__ void
execute(int *wholeNewWorld, const int *currWorld, const int *invaders, int nRows, int nCols, int eachThreadWork) {
    int threadId = (threadIdx.z * blockDim.y * blockDim.x + threadIdx.x * blockDim.y + threadIdx.y) +
                   (blockDim.x * blockDim.y * blockDim.z) *
                   (blockIdx.x * gridDim.y + blockIdx.y + blockIdx.z * gridDim.x * gridDim.y);
    int numTasksFinished = threadId * eachThreadWork;
    int currentRowIndex = numTasksFinished / nCols; //<-Init Row Index
    int currentColumnIndex = numTasksFinished % nCols; //<-Init Col Index
    for (int i = 0; i < eachThreadWork; i++) {
        //Check whether done
        if (currentRowIndex >= nRows or currentColumnIndex >= nCols) return;

        /*** ALGO PART ***/
        bool diedDueToFighting;
        int nextState = getNextState(currWorld, invaders, nRows, nCols, currentRowIndex, currentColumnIndex,
                                     &diedDueToFighting);
        setValueAt(wholeNewWorld, nRows, nCols, currentRowIndex, currentColumnIndex, nextState);
        if (diedDueToFighting) death[threadId]++;
        diedDueToFighting = false;
        /*** ALGO PART ***/

        //Get next rowIndex and colIndex
        if (currentColumnIndex == nCols - 1) {
            currentRowIndex++;
            currentColumnIndex = 0;
        }else{
            currentColumnIndex++;
        }
    }
}

/**
 * The main simulation logic.
 *
 * goi does not own startWorld, invasionTimes or invasionPlans and should not modify or attempt to free them.
 * nThreads is the number of threads to simulate with. It is ignored by the sequential implementation.
 */
int goi_cuda(int GRID_X, int GRID_Y, int GRID_Z, int BLOCK_X, int BLOCK_Y, int BLOCK_Z, int nGenerations,
             const int *startWorld, int nRows, int nCols, int nInvasions, const int *invasionTimes,
             int **invasionPlans) {
    // death toll due to fighting
    int deathToll = 0;
    int num = GRID_X * GRID_Y * GRID_Z * BLOCK_X * BLOCK_Y * BLOCK_Z;

    int start[MAX_THREAD_NUM];
    int host_death[MAX_THREAD_NUM];
    for (int i = 0; i < num; i++) {
        start[i] = 0;
    }

    hipError_t rc = hipMemcpyToSymbol(HIP_SYMBOL(death), &start, sizeof(start));

    if (rc != hipSuccess) {
        printf("Could not copy to device. Reason: %s\n", hipGetErrorString(rc));
    }
//    int death[num];
//    int* deathNum;

    // init the world!
    // we make a copy because we do not own startWorld (and will perform free() on world)
    int *world = static_cast<int *>(malloc(sizeof(int) * nRows * nCols));
    if (world == NULL) {
        return -1;
    }
    for (int row = 0; row < nRows; row++) {
        for (int col = 0; col < nCols; col++) {
            GlobalsetValueAt(world, nRows, nCols, row, col, GlobalgetValueAt(startWorld, nRows, nCols, row, col));
        }
    }

    int *worldCuda;
    hipMalloc((void **) &worldCuda, sizeof(int) * nRows * nCols);
    hipMemcpy(worldCuda, world, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);

#if PRINT_GENERATIONS
    printf("\n=== WORLD 0 ===\n");
    printWorld(world, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
    exportWorld(world, nRows, nCols);
#endif

    // Begin simulating
    int invasionIndex = 0;
    for (int i = 1; i <= nGenerations; i++) {
        // is there an invasion this generation?
        int *inv = NULL;
        int *invCuda = NULL;
        if (invasionIndex < nInvasions && i == invasionTimes[invasionIndex]) {
            // we make a copy because we do not own invasionPlans
            inv = static_cast<int *>(malloc(sizeof(int) * nRows * nCols));
            hipMalloc((void **) &invCuda, sizeof(int) * nRows * nCols);
            if (inv == NULL) {
                free(world);
                hipFree(worldCuda);
                return -1;
            }
            for (int row = 0; row < nRows; row++) {
                for (int col = 0; col < nCols; col++) {
                    GlobalsetValueAt(inv, nRows, nCols, row, col,
                                     GlobalgetValueAt(invasionPlans[invasionIndex], nRows, nCols, row, col));
                }
            }
            hipMemcpy(invCuda, inv, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
            invasionIndex++;
        }

        // create the next world state
//        int *wholeNewWorld = static_cast<int *>(malloc(sizeof(int) * nRows * nCols));
//        if (wholeNewWorld == NULL)
//        {
//            if (inv != NULL)
//            {
//                free(inv);
//                hipFree(invCuda);
//            }
//            free(world);
//            hipFree(worldCuda);
//            return -1;
//        }


//        hipMalloc((void**)&deathNum, num);
//        hipMemcpy(deathNum, death, num, hipMemcpyHostToDevice);
//
//        printf("HAHA\n");
//        printWorld(world,  nRows,  nCols);
//        printWorld(wholeNewWorld,  nRows,  nCols);
        int *wholeNewWorldCuda = NULL;
        hipMalloc((void **) &wholeNewWorldCuda, sizeof(int) * nRows * nCols);
//        for (int row = 0; row < nRows; row++)
//        {
//            for (int col = 0; col < nCols; col++)
//            {
//                GlobalsetValueAt(wholeNewWorld, nRows, nCols, row, col, 0);
//            }
//        }
//        hipMemcpy(wholeNewWorldCuda, wholeNewWorld, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
        dim3 gridDim(GRID_X, GRID_Y, GRID_Z);
        dim3 blockDim(BLOCK_X, BLOCK_Y, BLOCK_Z);
        int eachThreadWork = (nRows * nCols - 1) / num + 1;
        execute<<<gridDim, blockDim>>>(wholeNewWorldCuda, worldCuda, invCuda, nRows, nCols, eachThreadWork);
        hipDeviceSynchronize();

//     hipMemcpy(wholeNewWorld, wholeNewWorldCuda, sizeof(int) * nRows * nCols, hipMemcpyDeviceToHost);

//        hipMemcpy(death, deathNum, num, hipMemcpyDeviceToHost);
        // get new states for each cell
//        for (int row = 0; row < nRows; row++)
//        {
//            for (int col = 0; col < nCols; col++)
//            {
//                bool diedDueToFighting;
//                int nextState = getNextState(world, inv, nRows, nCols, row, col, &diedDueToFighting);
//                setValueAt(wholeNewWorld, nRows, nCols, row, col, nextState);
//                if (diedDueToFighting)
//                {
//                    deathToll++;
//                }
//            }
//        }

        if (inv != NULL) {
            free(inv);
            hipFree(invCuda);
        }

        // swap worlds
//        free(world);
        hipFree(worldCuda);
        worldCuda = wholeNewWorldCuda;
//        world = wholeNewWorld;

#if PRINT_GENERATIONS
        printf("\n=== WORLD %d ===\n", i);
        printWorld(world, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
        exportWorld(world, nRows, nCols);
#endif
    }

    rc = hipMemcpyFromSymbol(&host_death, HIP_SYMBOL(death), sizeof(start));

    if (rc != hipSuccess) {
        printf("Could not copy from device. Reason: %s\n", hipGetErrorString(rc));
    }

    for (int i = 0; i < num; i++) {
        deathToll += host_death[i];
    }

//    free(world);
    hipFree(worldCuda);
    return deathToll;
}
